#include "hip/hip_runtime.h"
#include "rotate_nms.cuh"
#include "rotate_nms_utils.h"
#include <vector>
#include <iostream>

#define DIVUP(m,n) ((m + n - 1)/n)
int64_t const threadsPerBlock = sizeof(unsigned long long) * 8;

__global__ void testKernel(float* val)
{
   int idx = threadIdx.x;
   val[idx] += 1;
   printf("current idx: %d", idx);
}

__device__ inline float devIoU(float const * const a, float const * const b) {
    float left = max(a[0], b[0]), right = min(a[2], b[2]);
    float top = max(a[1], b[1]), bottom = min(a[3], b[3]);
    float width = max(right - left + 1, 0.f), height = max(bottom - top + 1, 0.f);
    float interS = width * height;
    float Sa = (a[2] - a[0] + 1) * (a[3] - a[1] + 1);
    float Sb = (b[2] - b[0] + 1) * (b[3] - b[1] + 1);
    return interS / (Sa + Sb - interS);
  }


template <typename T>
__global__ void nms_rotated_cuda_kernel(
    const T* dev_boxes,
    const int n_boxes,
    const float iou_threshold,
    unsigned long long *mask_dev_ptr)
{
    const int row_start = blockIdx.y;
    const int col_start = blockIdx.x;

    const int row_size =
        min(n_boxes - row_start * threadsPerBlock, threadsPerBlock);
    const int col_size =
        min(n_boxes - col_start * threadsPerBlock, threadsPerBlock);
    
    __shared__ T block_boxes[threadsPerBlock*5];

    if(threadIdx.x < col_size)
    {
        block_boxes[threadIdx.x * 5 + 0] =
            dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 5 + 0];
        block_boxes[threadIdx.x * 5 + 1] =
            dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 5 + 1];
        block_boxes[(threadIdx.x * 5) + 2] =
            dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 5 + 2];
        block_boxes[threadIdx.x * 5 + 3] =
            dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 5 + 3];
        block_boxes[threadIdx.x * 5 + 4] =
            dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 5 + 4];
    }

    __syncthreads();

    if (threadIdx.x < row_size) {
        const int cur_box_idx = threadsPerBlock * row_start + threadIdx.x;
        const T* cur_box = dev_boxes + cur_box_idx * 5;
        int i = 0;
        unsigned long long t = 0;
        int start = 0;
        if (row_start == col_start) 
        {
            start = threadIdx.x + 1;
        }

        for (i = start; i < col_size; i++) 
        {
            if (single_box_iou_rotated<T>(cur_box, block_boxes + i * 5) > iou_threshold)  //
            {
                t |= 1ULL << i;
            }
        }

        const int col_blocks = DIVUP(n_boxes,threadsPerBlock);
        mask_dev_ptr[cur_box_idx * col_blocks + col_start] = t;

        if(row_start == 0 && col_start == 0)
        {
            if(threadIdx.x == 1)
            {
                // T iou = single_box_iou_rotated<T>(cur_box, block_boxes + 11 * 5);
                // printf("%f %f %f %f %f \n",cur_box[0],cur_box[1],cur_box[2],cur_box[3],cur_box[4]); //  
                // printf("%f %f %f %f %f \n",*(block_boxes + 11 * 5 + 0),*(block_boxes + 11 * 5 + 1),*(block_boxes + 11 * 5 + 2),*(block_boxes + 11 * 5 + 3),*(block_boxes + 11 * 5 + 4));
                // printf(" %llu\n",t);
                // printf(" %llu\n",mask_dev_ptr[cur_box_idx * col_blocks + col_start]);
                // printf(" %d\n",cur_box_idx * col_blocks + col_start);
                // printf("iou %f\n",iou);
            }
        }
    }
}

__global__ void print_mask(unsigned long long* mask, int box_nums, int cols)
{
    int idx = threadIdx.x;
    printf("enter print mask");

    if(idx < box_nums)
    {
        for(int i = 0; i < cols; i++)
        {

        }
    }
}

void test_kernel()
{
    printf("enter kernel\n");

    float test_host[5] = {0};
    
    float *test_dev = NULL;
    HANDLE_ERROR(hipMalloc((void **)&test_dev, 5 * sizeof(float)));
    HANDLE_ERROR(hipMemcpy(test_dev, test_host, 5 * sizeof(float),hipMemcpyHostToDevice));

    testKernel<<<1,5>>>(test_dev);
    
    HANDLE_ERROR(hipMemcpy(test_host,test_dev, 5 * sizeof(float), hipMemcpyDeviceToHost));

    for(int i = 0; i < 5; i++)
    {
        std::cout << test_host[i] << std::endl;
    }
}

void nms_rotated_cuda(const float *dets_host_ptr, float *dets_dev_ptr, std::vector<int>& ids, const int max_nms_input_nums, const int box_nums, const float iou_threshold, const int top_k)
{
    if(!dets_dev_ptr)
    {
        HANDLE_ERROR(hipMalloc((void **)&dets_dev_ptr, max_nms_input_nums * 5 * sizeof(float)));
    }
    HANDLE_ERROR(hipMemcpy(dets_dev_ptr, dets_host_ptr, box_nums*5*sizeof(float), hipMemcpyHostToDevice));


    const int col_blocks = DIVUP(box_nums,threadsPerBlock);
    dim3 blocks(col_blocks,col_blocks);
    dim3 threads(threadsPerBlock);


    unsigned long long *mask_host_ptr = new unsigned long long[box_nums * col_blocks];    
    unsigned long long *mask_dev_ptr = NULL;
    HANDLE_ERROR(hipMalloc((void **)&mask_dev_ptr,sizeof(unsigned long long) * box_nums * col_blocks));
    hipMemset(mask_dev_ptr, 0, sizeof(unsigned long long) * box_nums * col_blocks);
    
    nms_rotated_cuda_kernel<float><<<blocks,threadsPerBlock>>>(dets_dev_ptr,box_nums,iou_threshold,mask_dev_ptr);
    
    //hipDeviceSynchronize();
    // hipMemcpy is a tongbu function , so no need hipDeviceSynchronize
    HANDLE_ERROR(hipMemcpy(mask_host_ptr,mask_dev_ptr, sizeof(unsigned long long)* box_nums * col_blocks, hipMemcpyDeviceToHost)); //* box_nums * col_blocks

    std::vector<unsigned long long> remv(col_blocks);
    memset(&remv[0], 0ULL, sizeof(unsigned long long) * col_blocks);
    int num_to_keep = 0;

    // for(int i = 0; i < box_nums; i++)
    // {
    //     int cur = i * col_blocks; 
    //     for(int j = 0; j < col_blocks; j++)
    //     {
    //         std::cout << mask_host_ptr[cur+j] << " ";
    //     }
    //     std::cout << std::endl;
    // }

    bool break_flag = 0;
    for(int i = 0; i < box_nums; ++i)
    {
        int nblock = i / threadsPerBlock;
        int inblock = i % threadsPerBlock;

        // if(i == 2)
        // {
        //     std::bitset<64> bs(remv[nblock]);
        //     std::cout << bs << std::endl;
        // }

        if(!(remv[nblock] & (1ULL << inblock)))
        {
            ++num_to_keep;
            ids.push_back(i);
            unsigned long long* p = mask_host_ptr + i * col_blocks;
            for (int j = nblock; j < col_blocks; ++j) 
            {
                remv[j] |= p[j];
                if(num_to_keep == top_k)
                {
                    break_flag = 1;
                    break;
                }
            }
        }

        if(break_flag)
        {
            break;
        }
    }

    // for(int i = 0; i < ids.size(); i++)
    // {
    //     for(int j = 0; j < 5; j++)
    //     {
    //         std::cout << dets_host_ptr[ids[i] * 5 + j] << " " ; // << continuous_output[nms_ids[i] * 5 + j] << " ";
    //     }
    //     std::cout  << ids[i] << std::endl;
    // }

    delete mask_host_ptr;
    HANDLE_ERROR(hipFree(mask_dev_ptr));
}
// 4 9
// 4 10